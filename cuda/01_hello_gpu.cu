
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU (int n)
{
	printf("Hello from GPU with grid %d and thread %d\n", n, threadIdx.x);
	//printf("From:%d, %d ", n, blockIdx.x);
}

int main (void)
{
	helloFromGPU<<<1,10>>>(1);
	hipDeviceSynchronize();

	helloFromGPU<<<1,10>>>(2);
	hipDeviceSynchronize();

	printf("Hello CPU\n");

//	cudaDeviceSynchronize();
	
	return 0;
}
