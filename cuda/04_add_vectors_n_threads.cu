
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

#define N   10

__global__ void add( int *a, int *b, int *c ) {
    /**
     * threadIdx.x contém o Id da thread (no bloco) a ser executada
     */

	int tid = threadIdx.x;    // this thread handles the data at its thread id
    if (tid < N)
        c[tid] = a[tid] + b[tid];
}

int main( void ) {
    int a[N], b[N], c[N];
    int *dev_a, *dev_b, *dev_c;

    // allocate the memory on the GPU
    hipMalloc( (void**)&dev_a, N * sizeof(int));
    hipMalloc( (void**)&dev_b, N * sizeof(int));
    hipMalloc( (void**)&dev_c, N * sizeof(int));

    // fill the arrays 'a' and 'b' on the CPU
    for (int i=0; i<N; i++) {
        a[i] = -i;
        b[i] = i * i;
    }

    // copy the arrays 'a' and 'b' to the GPU
    hipMemcpy( dev_a, a, N * sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy( dev_b, b, N * sizeof(int),hipMemcpyHostToDevice);

    /**
     *  N é o no. de threads a ser executada em paralelo pela GPU
     */
    add<<<1,N>>>(dev_a,dev_b,dev_c );

    // copy the array 'c' back from the GPU to the CPU
    hipMemcpy(c, dev_c, N * sizeof(int),hipMemcpyDeviceToHost);

    // display the results
    for (int i=0; i<N; i++) {
        printf( "[%d] %d + %d = %d\n",i, a[i], b[i], c[i] );
    }

    // free the memory allocated on the GPU
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

	getchar();

    return 0;
}
