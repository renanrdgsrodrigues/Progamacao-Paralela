
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

/**
 * Função que realiza a soma de a + b e retorna um inteiro.
 * __device__ apenas o dispositivo pode chamar esta função.
 */
__device__ int addem( int a, int b ) {
    return a + b;
}

/**
 * Função que vai ser chamada do host para o dispositivo.
 * realiza a soma de a + b e guarda o resultado no ponteiro passado
 */
__global__ void add( int a, int b, int *c ) {
    *c = addem( a, b );
}

int main(void){
    /** variavel de resposta **/	
    int c;			/** ponteiro a ser passado para as funções no dispositivo **/
    int *dev_c;		/** alocação de memória no dispositivo **/
	
    hipMalloc((void**)&dev_c, sizeof(int));
    
	/** chamada a função add para ser executada no dispositivo **/
    add<<<1,1>>>(2,7,dev_c);
	
    /** cópia do conteudo do ponteiro dev_c para a variavel c **/
    hipMemcpy(&c,dev_c, sizeof(int),hipMemcpyDeviceToHost);
	
    printf( "2 + 7 = %d\n", c );

    /** liberação de memoria alocada no dispositivo **/
    hipFree(dev_c);

	//getchar();
    return 0;
}
